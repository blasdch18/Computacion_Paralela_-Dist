
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include<iostream>

using namespace std;

// compute vector sum C = A+B
// each thread performs one pair-wise addition
__global__ // executed on the device, only callable from the host
void vecAddKernel(float *A, float *B, float *C, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i < n) {
		C[i] = A[i] + B[i];
	}
}

// Compute vector sum h_C = h_A + h_B
void vecAdd(float *h_A, float *h_B, float *h_C, int n) {
	 
	// calculate the sum with a simple for loop
	/***************************
	*for (int i = 0; i < 100; i++)
	*	h_C[i] = h_A[i] + h_B[i];
	****************************/

	int size = n*sizeof(float);
	float *d_A, *d_B, *d_C;

	//1. Allocate global memory on the device for A, B and C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// copy A and B to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	//2. Kernel launch code - to have the device to perform the actual vector addition
	// Kernel invocation with 256 threads

	vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

	//3. copy C from the device memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Free device vectors
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main(void) {

	// create and host vectors
	float *h_A, *h_B, *h_C;
	int n = 2000000;

	// allocate memory for host vectors
	h_A = (float*)malloc(sizeof(float)*n);
	h_B = (float*)malloc(sizeof(float)*n);
	h_C = (float*)malloc(sizeof(float)*n);
	
	// fill A and B host vectors with a simple for loop
	srand(time(NULL));
	for (int i = 0; i < n; i++) {
		h_A[i] = ((((float)rand() / (float)(RAND_MAX)) * 100));
		h_B[i] = ((((float)rand() / (float)(RAND_MAX)) * 100));
	}
    cout<<"\n\nvectorA=";
    for (int i = 0; i < n; i++) {
		cout<< h_A[i] << " " ;
	}
    cout<<"\n\n\nvectorB=";
    for (int i = 0; i < n; i++) {
		cout<< h_B[i] ;
	}
	// call vecAdd to compute vector sum
	vecAdd(h_A, h_B, h_C, n);
    cout<<"\n\n\nvectorC=";
    for (int i = 0; i < n; i++) {
		cout<< h_C[i] ;
	}
	// check the result
	bool valid = true;
	for (int i = 0; i < n; i++) {
		if (h_A[i] + h_B[i] != h_C[i])
			valid = false;
			break;
	}

	if (!valid)
		printf("the result is not correct\n!");
	else
		printf("ok sum completed with success!\n");

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);


	return 0;
}